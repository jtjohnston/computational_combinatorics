
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<fstream>
#include<algorithm>

// This value is the largest subsets we must unrank.
// This value must be set before compiling because the size of arrays
// must be fixed (not dynamic) on the GPU.
#define LARGEST_SUBSET 10

using namespace::std;

// Device code to compute the binomial coefficient "n choose k."
// This code is patterned off of: http://stackoverflow.com/a/3025547/4187033
// This function is callable from the device (GPU) and runs on the device (GPU).
__device__ long binom_d(int n, int k){
	long ntok = 1;
	long ktok = 1;
	if(k > n){
		return 0;
	}
	int i, j;
	j = n;
	for(i=1; i<=min(k, n-k); i++){
		ntok *= j;	//A falling factorial (top), n (n-1) (n-2) ... 
		ktok *= i;	//A rising factorial (bottom), 1 (2) (3) ...
		j--;
	}
	return ntok / ktok;
}

// The code is identical to binom_d except that it runs on the CPU (instead of GPU).
long binom_h(int n, int k){
	long ntok = 1;
	long ktok = 1;
	if(k > n){
		return 0;
	}
	int i, j;
	j = n;
	for(i=1; i<=min(k, n-k); i++){
		ntok *= j;
		ktok *= i;
		j--;
	}
	return ntok / ktok;
}



// This code generates a k-subset of an n-element set [0, 1, ..., n-1] and stores it in Kset.
// It finds the "initial_value" k-subset in lexicographic order.
// This code is called from the device (GPU) and runs on the device (GPU).
__device__ void unrank_combination(int n, int k, long initial_value, int* Kset) {
	long cash_on_hand = initial_value;
	int digit;
	long cost_to_increment;
	Kset[0] = 0;	//Initialize the first element.
					//Each of the following elements will start off one bigger than the previous element.
	//Use the cash_on_hand value to "pay" for incrementing each digit.
	//Pay 1-unit for each combination that is "skipped" over.
	//E.g. To increment the 0 in 0, 1, 2, ..., k-1 to a 1 (and force the others to increment to 2, 3, ..., k)
	//it would cost binom(n-1, k-1) since we skipped over each combination of the form
	// 0 * * * ... * and there are binom(n-1, k-1) of those combinations
	for(digit=0; digit<k-1; digit++){
		//There are n-1-Kset[digit] elements left to choose from.
		//Those elements must be used to fill k-1-digit places.
		cost_to_increment = binom_d( n-1-Kset[digit], k-1-digit );
		while(cost_to_increment <= cash_on_hand){
			Kset[digit]++;
			cash_on_hand = cash_on_hand - cost_to_increment;
			cost_to_increment = binom_d( n-1-Kset[digit], k-1-digit );
		}
		Kset[digit+1] = Kset[digit]+1;	//Ititialize the next element of Kset making sure the elements
										//come in sorted order.
	}
	//Kset[k-1] has been initialized to Kset[k-2]+1 (last step).
	//Now, if there is anything left to pay, we simply increment Kset[k-1] by this amount.
	Kset[k-1] += cash_on_hand;
}

// This code is identical to unrank_combination except that it is called by the CPU and runs on CPU.
// The code was originally intended for debugging, but is also helpful for displaying individual subsets of interest.
void unrank_combination_h(int n, int k, long initial_value, int* Kset) {
	long cash_on_hand = initial_value;
	int digit;
	long cost_to_increment;
	Kset[0] = 0;	//Initialize the first element.
					//Each of the following elements will start off one bigger than the previous element.
	//Use the cash_on_hand value to "pay" for incrementing each digit.
	//Pay 1-unit for each combination that is "skipped" over.
	//E.g. To increment the 0 in 0, 1, 2, ..., k-1 to a 1 (and force the others to increment to 2, 3, ..., k)
	//it would cost binom(n-1, k-1) since we skipped over each combination of the form
	// 0 * * * ... * and there are binom(n-1, k-1) of those combinations
	for(digit=0; digit<k-1; digit++){
		//There are n-1-Kset[digit] elements left to choose from.
		//Those elements must be used to fill k-1-digit places.
		cost_to_increment = binom_h( n-1-Kset[digit], k-1-digit );
		while(cost_to_increment <= cash_on_hand){
			Kset[digit]++;
			cash_on_hand = cash_on_hand - cost_to_increment;
			cost_to_increment = binom_h( n-1-Kset[digit], k-1-digit );
		}
		Kset[digit+1] = Kset[digit]+1;	//Ititialize the next element of Kset making sure the elements
										//come in sorted order.
	}
	//Kset[k-1] has been initialized to Kset[k-2]+1 (last step).
	//Now, if there is anything left to pay, we simply increment Kset[k-1] by this amount.
	Kset[k-1] += cash_on_hand;
}



// This function is the heart of the search.
// When this code is called, many threads are spawned on the GPU.
// Each thread is assigned a number (my_subset).
// Each thread unranks the subset assigned by my_subset, and determines whether or not it is a potential switching set.
// If the induced subgraph (on the specific subset of vertices) is regular AND
// If every vertex outside the subraph is adjacent to either 0, 1/2 or all vertices in the subgraph AND
// If there exists a vertex adjacent to exactly 1/2 of all vertices in the subgraph,
// Then, the subset of vertices is reported as a potential switching set.

__global__ void examine_subsets(int n, int k, long MAX, short* A, short* Results, long offset){
	const long my_subset = threadIdx.x + blockIdx.x*blockDim.x + offset;
	const int my_index = threadIdx.x + blockIdx.x*blockDim.x;
	if( my_subset < MAX ){		//MAX = number of subsets to examine.  
								//We make this check because more threads may be spawned then there are subsets that need to be checked.
		int i, j;
		int Kset[LARGEST_SUBSET];	//Kset will store the indices of the vertices this thread will look at.
		for(i=0; i<k; i++){
			Kset[i] = 0;
		}

		//urank_combination modifies Kset to be the specified vertices that should be examined by this thread.
		unrank_combination(n, k, my_subset, Kset);
		
		//induced_subgraph is the adjacency matrix of the subgraph induced by the vertices in Kset.
		//All 2D matrices are stored in 1D form--i.e. entry ij of an (m x n) matrix is in position i*N+j.
		short induced_subgraph[LARGEST_SUBSET*LARGEST_SUBSET];
		for(i=0; i<k; i++){
			for(j=0; j<k; j++){
				induced_subgraph[i*k + j] = A[ Kset[i]*n + Kset[j] ];	//A is n by n (in 1D form) but local_A is k by k (in 1D form)
			}
		}

		//Now, we need to check if induced_subgraph is a potential switching set.
		//First, we need to check if it is regular...
		//To do this, we check if every row sum is the same.
		Results[my_index] = 1;		//Initialize to believing this is a switching set.
									//We change this to zero if we find any reason to know that it isn't one.
		short first_row_sum = 0;
		short this_row_sum = 0;
		for(j=0; j<k; j++){
			first_row_sum += induced_subgraph[j];
		}
		for(i=1; i<k; i++){
			this_row_sum = 0;
			for(j=0; j<k; j++){
				this_row_sum += induced_subgraph[i*k + j];
			}
			if(this_row_sum != first_row_sum){
				Results[my_index] = -1;			//Not regular, hence, not a switching set.
			}
		}	//End check for subgraph regularity.
		

		//If the graph is regular, check to see if it is a switching candidate.
		//It is NOT switching if any vertex (outside of Kset) is not adjacent to either 0, 1/2, or all of these vertices.
		//It is also NOT switching if there isn't at least one vertex (outside of Kset) adjacent to 1/2 of these vertices.
		if(Results[my_index]==1){
			Results[my_index] = 2;		//This flag notes that we have not yet found a vertex adjacent to 1/2.
										//If this is not 0 and we find a vertex adjacent to 1/2, then it becomes a 1.
										//If this ever becomes 0, it stays 0.
			for(i=0; i<n; i++){
				//How many vertices in Kset is vertex i adjacent to?
				//To save registers, recycle first_row_sum for the number of vertices seen and this_row_sum as an indicator for whether or not i is in Kset.
				first_row_sum = 0;
				this_row_sum = 0;
				for(j=0; j<k; j++){
					first_row_sum += A[ i*n + Kset[j] ];		// i'th row, column Kset[j]... 
																// 1 if vertex i is adjacent to Kset[j] (the j'th element of our potential switching set).
					if(i == Kset[j]){
						this_row_sum = 1;						// This indicates that vertex i is in the Kset (and should not be considered).
					}
				}
				if(this_row_sum == 0){	//this_row_sum indicates whether or not vertex i is in Kset.
					//i is not in Kset (since the indicator is 0).
					//Check to see if i is adjacent to 0, 1/2 or all of Kset.
					if( (first_row_sum != 0) && (first_row_sum != k) && (first_row_sum != (k/2)) ){
						Results[my_index]=0;
					}else{
						if( (first_row_sum == (k/2)) && Results[my_index]==2){
							Results[my_index]=1;
						}
					}
				}
			}
		}//End if(Results[my_index]==1) -- the check for outside vertices adjacent to 0, 1/2 or all of Kset
	}//End of if(my_subset < MAX)
}






int main(int argc, char** argv){
	int N = atoi( argv[1] );	//Number of vertices in the Johnson graph;
								//Should be the number of lines in matrix_NN_KK.txt (data file).
	int K = atoi( argv[2] );	//Size of the subset to inspect.
	
	int NN = atoi( argv[3] );	
	int KK = atoi( argv[4] );	//The vertices of the graph are KK-subsets of an NN-element set.
	int GPU = atoi( argv[5] );	//Which GPU to use (can set this to be 0 if you only have a single GPU).


	int* Kset;
	Kset = (int *) malloc( K*sizeof(int) );

	int* sKset;
	sKset = (int *) malloc( KK*sizeof(int) );

	char fileName[50];
	//Expect an adjaceny matrix in current working directory with name matrix_NN_KK.txt (where NN and KK are appropriately replaced).
	sprintf(fileName, "matrix_%d_%d.txt", NN, KK);


	short* h_A;		//A pointer to host memory where the adjacency matrix will be stored
	short* d_A;		//A pointer to device (GPU) memory where the adjaceny matrix will be stored on the GPU
	short* h_Results;	//A list of indicators (potential switching or not) stored on host
	short* d_Results;	//A list of indicators (potential switching or not) stored on the device (GPU).

	const long BATCH = 1000000000;	//The number of threads to spawn at one time; it is also the amount of memory (approx) that will be allocated on the GPU.

	long number_of_subsets = binom_h(N-1, K-1);		//Because of vertex transitivity, only check subsets containing vertex 0.
													//If the graph is not vertex transitive, this should change to binom_h(N, K).

	printf("Checking %ld subsets.\n", number_of_subsets);	//Debugging, could be deleted.
	long offset;

	long size_of_subsets = BATCH*sizeof( short );	//Amount of memory that will be allocated for h_Results and d_Results.
	long size_of_A = N*N*sizeof(short);				//Amount of memory allocated to store the adjacency matrix.

	int i, j, k;	//indices
	h_A = (short *) malloc( size_of_A );				//allocate memory on host (CPU) for adjacency matrix
	h_Results = (short *) malloc( size_of_subsets );	//allocate memory on host (CPU) for results (indicator array)
	

	//Read in the adjacency matrix from the file.
	ifstream fin;
	fin.open(fileName);
	for(i=0; i<N; i++){
		for(j=0; j<N; j++){
			fin >> h_A[i*N + j];	//Again, storing in 1D form.  Entry ij of an m x n matrix appears at entry i*n + j.
		}
	}
	fin.close();
	

	for(i=0; i<BATCH; i++){
		h_Results[i]=0;				//Initialize the results array (CPU-side).
	}


	hipSetDevice( GPU );								//Select which GPU to use.
	hipMalloc((void **) &d_A, size_of_A);				//allocate memory for the adjacency matrix on the GPU
	hipMalloc((void **) &d_Results, size_of_subsets);	//allocate memory for the results (indicator array) on the GPU


	hipMemcpy(d_A, h_A, size_of_A, hipMemcpyHostToDevice);	//Copy the adjacency matrix from the CPU to the GPU
	hipMemcpy(d_Results, h_Results, size_of_subsets, hipMemcpyHostToDevice);	//Copy the Results (only initialized to 0) to the GPU.
	printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));		//Check for errors--debugging only (could be deleted).


	for(offset=0; offset<=number_of_subsets; offset+=BATCH){
		printf("Examining subsets %ld to %ld.\n", offset, offset+BATCH-1);
		
		//Invoke the kernel that inspects subsets.
		//IMPORTANT NOTE: If BATCH size is changed, it is important that examine_subsets<<< a, b>>>(...) satisfy a*b=BATCH
		//and keep in mind that BATCH/1000 is integer arithmetic (not floating point).
		//Depending on GPU architecture, it may provide a performance boost to make BATCH a power of 2 and then a=BATCH/b where b = 16, 32, or 64.
		examine_subsets<<< BATCH/1000, 1000 >>>(N, K, number_of_subsets, d_A, d_Results, offset);
		
		printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));		//Check for errors--debugging only (could be deleted).
																				//This only catches kernel invocation errors since it occurs before the cudaDeviceSynchronize().

		hipMemcpy(h_Results, d_Results, size_of_subsets, hipMemcpyDeviceToHost);	//Copy results indicator from GPU back to CPU.
		hipDeviceSynchronize();	//Prevents the CPU code from moving on to the next for-loop before the execution on the GPU (and subsequent memory copy) finishes.

		for(i=0; (i<BATCH) && (i+offset<number_of_subsets) ; i++){
			if( h_Results[i] == 1 ){	//Found a potential switching set.
										//Print off the candidate switching set.
				for(j=0; j<K; j++){
					Kset[j]=0;
				}
				unrank_combination_h(N, K, i+offset, Kset);
				printf("Possible switching set %ld: ", i+offset);
				for(j=0; j<K; j++){
					unrank_combination_h(NN, KK, Kset[j], sKset);
					printf("{ ");
					for(k=0; k<KK; k++){
						printf("%d ", sKset[k]);
					}
					printf("} ");
				}
				printf("\n");
			}
		}
	}
	printf("Done.\n");

	free(h_A);
	free(h_Results);
	hipFree(d_A);
	hipFree(d_Results);
	
	return 0;
}
